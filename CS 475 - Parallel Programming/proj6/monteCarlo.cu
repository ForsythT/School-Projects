#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#define OUTPUT_FILE		"results.txt"

#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// helper functions

float
Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}


__global__ void MonteCarlo( float *xcs, float *ycs, float *rs, float *numHits ) {
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	// monte carlo simulation

	float xc = xcs[gid];
	float yc = ycs[gid];
	float  r =  rs[gid];

	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;

	if ( d < 0.) {
		return;
	}

	d = sqrt( d );
	float t1 = ( -b + d ) / ( 2.*a );
	float t2 = ( -b - d ) / ( 2.*a );
	float tmin = t1 < t2 ? t1 : t2;

	if ( tmin < 0. ) {
		return;
	}

	float xcir = tmin;
	float ycir = tmin;

	float nx = xcir - xc;
	float ny = ycir - yc;
	float n = sqrt( nx*nx + ny*ny );
	nx /= n;
	ny /= n;

	float inx = xcir - 0.;
	float iny = ycir - 0.;
	float in = sqrt( inx*inx + iny*iny );
	inx /= in;
	iny /= in;

	float dot = inx*nx + iny*ny;
	float outx = inx - 2.*nx*dot;
	float outy = iny - 2.*ny*dot;

	float t = ( 0. - ycir ) / outy;

	if ( t < 0. ) {
		return;
	}
	
	prods[tnum] = 1;

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		numHits[wgNum] = prods[0];
}

// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	TimeOfDaySeed( );

	// allocate host memory:

	float * xcs = new float [ SIZE ];
	float * ycs = new float [ SIZE ];
	float * rs = new float [ SIZE ];
	float * numHits = new float [ SIZE/BLOCKSIZE ];

	// fill the random-value arrays:
        for( int n = 0; n < SIZE; n++ )
        {       
                xcs[n] = Ranf( XCMIN, XCMAX );
                ycs[n] = Ranf( YCMIN, YCMAX );
                rs[n] = Ranf(  RMIN,  RMAX ); 
        } 

	// allocate device memory:

	float *dA, *dB, *dC, *dD;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE, 1, 1 );
	dim3 dimsD( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dD), (SIZE/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dA, xcs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, ycs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dC, rs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        MonteCarlo<<< grid, threads >>>( dA, dB, dC, dD );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Block Size = %10d, Array Size = %10d, MegaTrials/Second = %10.2lf\n", BLOCKSIZE, SIZE, megaMultsPerSecond );

	// set performance in results text file

	FILE *f = fopen(OUTPUT_FILE, "a");
	if (f != NULL) {
		fprintf(f, "%f\t", megaMultsPerSecond);
	} else {
		printf("Can not open file.\n");
	}	
	
	// copy result from the device to the host:

	status = hipMemcpy( numHits, dD, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the numHits :
	double sum = 0.;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		sum += (double)numHits[i];
	}
	printf( "\nnumHits = %10.2lf\n", sum );
	double arraySize = SIZE;
	double probability = sum / arraySize;
	printf( "\nprobability = %10f\n", probability );

	// clean up memory:
	delete [ ] xcs;
	delete [ ] ycs;
	delete [ ] rs;
	delete [ ] numHits;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );
	status = hipFree( dD );
		checkCudaErrors( status );


	return 0;
}

